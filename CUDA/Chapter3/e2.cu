#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <iostream>
#include <chrono>

using namespace std;

/*MatVecMul_Kernel*/
__global__
void MatVecMul_Kernel(float* A, float* B, float* C, int n) {
	int i = threadIdx.x;
	int offset;
	float sum = 0;
	if (i < n) {
		for (int j = 0; j < n; j++) {
			offset = i*n + j;
			sum += A[offset] * B[j];
		}
		C[i] = sum;
	}
}

/*MatVecMul_GPU*/
void MatVecMul_GPU(float* h_A, float* h_B, float* h_C, int n) {
	int sizeM = n*n * sizeof(float);
	int sizeV = n * sizeof(float);
	float *d_A;
	float *d_B;
	float *d_C;
	hipMalloc(&d_A, sizeM);
	hipMemcpy(d_A, h_A, sizeM, hipMemcpyHostToDevice);
	hipMalloc(&d_B, sizeV);
	hipMemcpy(d_B, h_B, sizeV, hipMemcpyHostToDevice);
	hipMalloc(&d_C, sizeV);
	hipMemcpy(d_C, h_C, sizeV, hipMemcpyHostToDevice);
	//dim3 dimGrid(ceil(n / 32.0), 1, 1);
	//dim3 dimBlock(32.0, 1, 1);
	MatVecMul_Kernel <<< 1, 10 >>> (d_A, d_B, d_C, n);
	hipMemcpy(h_C, d_C, sizeV, hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

int main() {
	//Host Matrix
	float *h_A, *h_B, *h_C;

	int n = 10;

	h_A = (float*)malloc(n*n * sizeof(float));
	h_B = (float*)malloc(n * sizeof(float));
	h_C = (float*)malloc(n * sizeof(float));

	//Create Matrix
	for (int i = 0; i < n*n; i++) {
		h_A[i] = 1.0;
	}

	//Create Vector
	for (int i = 0; i < n; i++) {
		h_B[i] = 1.0;
		h_C[i] = 1.0;
	}

	//MatVecMul (Main)
	chrono::time_point<chrono::system_clock> MatVecMul_GPU_Start, MatVecMul_GPU_End;
	MatVecMul_GPU_Start = chrono::system_clock::now();
	MatVecMul_GPU(h_A, h_B, h_C, n);
	MatVecMul_GPU_End = chrono::system_clock::now();

	cout << "MatVecMul_GPU: " << chrono::duration_cast<chrono::nanoseconds>(MatVecMul_GPU_End - MatVecMul_GPU_Start).count() << "ns." << endl;

	//Print MatVecMul
	for (int i = 0; i < n; i++) {
			cout << h_C[i] << " ";
	}
	cout << endl;

	//Free
	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}
